#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <torch/library.h>
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <hip/hip_runtime.h>
#include <vector>
#include <iostream> // DEBUG ONLY

/* Based on the official PyTorch implementation of NMS using CUDA from:
https://github.com/pytorch/vision/blob/main/torchvision/csrc/ops/cuda/nms_kernel.cu
 * Official code was changed by Felix Sattler 2023 to include:
 * - Mean and variance calculations for every kept bounding boxes & scores over all overlapping candidates
 * - Coalesced memory accesses for the nms map kernel in CUDA & minor tweaks
 * - Modifications to compute the number of overlapping candidates per kept bounding box index
*/

// Hard-coded maximum. Increase if needed.
#define MAX_COL_BLOCKS 1000

#define DIVUP(m, n) (((m)+(n)-1) / (n))
#define PARENT_INDEX(x) (x - 1)

int64_t const threadsPerBlock = sizeof(unsigned long long) * 8;
int64_t const threadsPerBlockLinear = 256;


template<typename T, typename Ts>
__device__ inline bool devIoU(const T &a, const T &b, const float threshold) {
    Ts left = max(a.x, b.x), right = min(a.z, b.z);
    Ts top = max(a.y, b.y), bottom = min(a.w, b.w);
    Ts width = max(right - left + 1, (Ts) 0), height = max(bottom - top + 1, (Ts) 0);
    using acc_T = at::acc_type<Ts, /*is_cuda=*/true>;
    acc_T interS = (acc_T) width * height;
    acc_T Sa = ((acc_T) a.z - a.x + 1) * (a.w - a.y + 1);
    acc_T Sb = ((acc_T) b.z - b.x + 1) * (b.w - b.y + 1);
    return (interS / (Sa + Sb - interS)) > threshold;
}

template<typename T>
__global__ void nms_map_impl(const int64_t n_boxes,
                             const T nms_overlap_thresh,
                             const T *dev_boxes,
                             const int64_t *idx,
                             int64_t *dev_mask) {
    using Tvec = typename std::conditional<std::is_same<T, float>::value, float4, double4>::type;

    const int row_start = blockIdx.y;
    const int col_start = blockIdx.x;

    if (row_start > col_start)
        return;

    const int row_size =
            min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
    const int col_size =
            min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

    // We can coalesce this load into a single float4 or double4
    __shared__ Tvec block_boxes[threadsPerBlock];
    if (threadIdx.x < col_size) {
        block_boxes[threadIdx.x] = *reinterpret_cast<const Tvec *>(&dev_boxes[
                idx[(threadsPerBlock * col_start + threadIdx.x)] * 4]);
    }
    __syncthreads();

    if (threadIdx.x < row_size) {
        const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
        const Tvec cur_box = *reinterpret_cast<const Tvec *>(dev_boxes + idx[cur_box_idx] * 4);
        int i = 0;
        unsigned long long t = 0;
        int start = 0;
        if (row_start == col_start) {
            start = threadIdx.x + 1;
        }
        for (i = start; i < col_size; i++) {
            if (devIoU<Tvec, float>(cur_box, block_boxes[i]/* + i * 4*/, nms_overlap_thresh)) {
                t |= 1ULL << i;
            }
        }
        const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
        dev_mask[cur_box_idx * col_blocks + col_start] = t;
    }
}

/* The original code of this function was taken from:
 * https://github.com/gdlg/pytorch_nms/blob/master/src/nms_kernel.cu
 * Copyright (c) 2018, Grégoire Payen de La Garanderie, Durham University
 * It was modified by Felix Sattler, 2023 including the following modifications:
 * - index changed to avoid double visits of diagonal elements
 * - summing loop to collect how often a kept index is referenced
 */
__global__ void
nms_reduce_impl(const int boxes_num,
                const int col_blocks,
                int top_k,
                const int64_t *dets,
                const int64_t *idx,
                int64_t *keep,
                int64_t *parent_object_index,
                int64_t *parent_ref_count,
                int64_t *num_to_keep) {

    int64_t remv[MAX_COL_BLOCKS] = {0};
    int num_to_keep_ = 0;

    for (int i = 0; i < boxes_num; i++) {
        int nblock = i / threadsPerBlock;
        int inblock = i % threadsPerBlock;

        if (!(remv[nblock] & (1ULL << inblock))) {
            keep[num_to_keep_] = idx[i];

            const int64_t *p = &dets[0] + i * col_blocks;
            for (int j = nblock; j < col_blocks; j++) {
                remv[j] |= p[j];
            }

            // 'i + 1' since 'i' is visited below anyway (i.e. self-intersection)
            for (int j = i + 1; j < boxes_num; j++) {
                int nblockj = j / threadsPerBlock;
                int inblockj = j % threadsPerBlock;

                if (p[nblockj] & (1ULL << inblockj)) {
                    parent_object_index[idx[j]] = num_to_keep_ + 1;
                }
            }
            parent_object_index[idx[i]] = num_to_keep_ + 1;
            num_to_keep_++;

            if (num_to_keep_ == top_k)
                break;
        }
    }

    // Initialize the rest of the keep array to avoid uninitialized values.
    for (int i = num_to_keep_; i < boxes_num; ++i)
        keep[i] = 0;

    // collect the number of times each parent is referenced
    for (int i = 0; i < boxes_num; i++) {
        parent_ref_count[PARENT_INDEX(parent_object_index[i])] += 1;
    }

    *num_to_keep = min(top_k, num_to_keep_);
}


// We observed a bug on CUDA version 12.1 where the computation using __syncthreads() and only one thread to accumulate
// would not work anymore. On CUDA version 11.7 we successfully ran this code with no issues.
// Therefore, we currently provide a CPU implementation
#ifdef COMPUTE_MEAN_VAR_GPU
template<typename T>
__global__ void nms_mean_impl(const int64_t parent_object_num,
                              const T *dev_boxes,
                              const T *dev_scores,
                              const int64_t *parent_ref_index,
                              const int64_t *parent_ref_count,
                              T *mean_boxes,
                              T *mean_scores) {
    using Tvec = typename std::conditional<std::is_same<T, float>::value, float4, double4>::type;

    __shared__ Tvec mean_boxes_accm[threadsPerBlockLinear];  //local block memory cache
    __shared__ T mean_scores_accm[threadsPerBlockLinear];

    const int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i >= parent_object_num) {
        return;
    }

    T inv_N = static_cast<T>(1.0) / static_cast<float>(parent_ref_count[PARENT_INDEX(parent_ref_index[i])]);
    inv_N = isinf(inv_N) ? 0.0 : inv_N;

    // coalesced loads using float4 vector types
    const auto boxes = *reinterpret_cast<const Tvec *>(&dev_boxes[i * 4]);
    mean_boxes_accm[threadIdx.x] = {
            static_cast<T>(0.5) * (boxes.x + boxes.z) * inv_N,
            static_cast<T>(0.5) * (boxes.y + boxes.w) * inv_N,
            (boxes.z - boxes.x) * inv_N,
            (boxes.w - boxes.y) * inv_N
    };
    mean_scores_accm[threadIdx.x] = dev_scores[i] * inv_N;

    __syncthreads();

    // write (this is done by one thread)
    //FIXME There must be a way to do this more efficiently
    if (threadIdx.x == 0) {
        for (int j = 0; j < blockDim.x; j++) {
            const int k = j + blockIdx.x * blockDim.x;
            if (k < parent_object_num) {
                const int k_id = PARENT_INDEX(parent_ref_index[k]);
                auto mean = *reinterpret_cast<Tvec *>(&mean_boxes[k_id * 4]);
                mean = {mean.x + mean_boxes_accm[j].x,
                        mean.y + mean_boxes_accm[j].y,
                        mean.z + mean_boxes_accm[j].z,
                        mean.w + mean_boxes_accm[j].w};
                reinterpret_cast<Tvec *>(mean_boxes)[k_id] = mean;
                mean_scores[k_id] += mean_scores_accm[j];
            }
        }
    }
}


template<typename T>
__global__ void nms_var_impl(const int64_t parent_object_num,
                             const T *dev_boxes,
                             const T *dev_scores,
                             const int64_t *parent_ref_index,
                             const int64_t *parent_ref_count,
                             const T *mean_boxes,
                             const T *mean_scores,
                             T *variances) {
    using Tvec = typename std::conditional<std::is_same<T, float>::value, float4, double4>::type;

    __shared__ Tvec var_boxes_accm[threadsPerBlockLinear];  //local block memory cache
    __shared__ T var_scores_accm[threadsPerBlockLinear];

    const int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i >= parent_object_num) {
        return;
    }

    const int i_id = PARENT_INDEX(parent_ref_index[i]);

    T inv_N = 1.0 / (static_cast<float>(parent_ref_count[i_id]) - 1.0);
    inv_N = isinf(inv_N) ? 0.0f : inv_N;

    const auto boxes = *reinterpret_cast<const Tvec *>(&dev_boxes[i * 4]);
    const auto mean = *reinterpret_cast<const Tvec *>(&mean_boxes[i_id * 4]);
    Tvec tmp = {mean.x - static_cast<T>(0.5) * (boxes.x + boxes.z),
                mean.y - static_cast<T>(0.5) * (boxes.y + boxes.w),
                mean.z - (boxes.z - boxes.x),
                mean.w - (boxes.w - boxes.y)};

    var_boxes_accm[threadIdx.x] = {tmp.x * tmp.x * inv_N,
                                   tmp.y * tmp.y * inv_N,
                                   tmp.z * tmp.z * inv_N,
                                   tmp.w * tmp.w * inv_N};
    const T tmp_delta = mean_scores[i_id] - dev_scores[i];
    var_scores_accm[threadIdx.x] = tmp_delta * tmp_delta * inv_N;

    __syncthreads();

    // write (this is done by one thread)
    if (threadIdx.x == 0) {
        for (int j = 0; j < blockDim.x; j++) {
            const int k = j + blockIdx.x * blockDim.x;
            if (k < parent_object_num) {
                const int k_id = PARENT_INDEX(parent_ref_index[k]) * 5;
                variances[k_id + 0] += var_boxes_accm[j].x;
                variances[k_id + 1] += var_boxes_accm[j].y;
                variances[k_id + 2] += var_boxes_accm[j].z;
                variances[k_id + 3] += var_boxes_accm[j].w;
                variances[k_id + 4] += var_scores_accm[j];
            }
        }
    }
}
#else

template<typename T>
struct vec4 {
    T x;
    T y;
    T z;
    T w;
};

template<typename T>
void nms_mean_impl_cpu(const int64_t parent_object_num,
                       const T *h_boxes,
                       const T *h_scores,
                       const int64_t *parent_ref_index,
                       const int64_t *parent_ref_count,
                       T *mean_boxes,
                       T *mean_scores) {

    for (int i = 0; i < parent_object_num; i++) {
        const int i_id = PARENT_INDEX(parent_ref_index[i]);

        T inv_N = static_cast<T>(1.0) / static_cast<float>(parent_ref_count[i_id]);
        inv_N = isinf(inv_N) ? 0.0 : inv_N;


        // coalesced loads using float4 vector types
        const auto boxes = *reinterpret_cast<const vec4<T> *>(&h_boxes[i * 4]);
        const vec4<T> mean_box = {
                static_cast<T>(0.5) * (boxes.x + boxes.z) * inv_N,
                static_cast<T>(0.5) * (boxes.y + boxes.w) * inv_N,
                (boxes.z - boxes.x) * inv_N,
                (boxes.w - boxes.y) * inv_N
        };
        const T mean_score = h_scores[i] * inv_N;

        auto mean = *reinterpret_cast<vec4<T> *>(&mean_boxes[i_id * 4]);
        mean = {mean.x + mean_box.x,
                mean.y + mean_box.y,
                mean.z + mean_box.z,
                mean.w + mean_box.w};
        reinterpret_cast<vec4<T> *>(mean_boxes)[i_id] = mean;
        mean_scores[i_id] += mean_score;
    }
}


template<typename T>
void nms_var_impl_cpu(const int64_t parent_object_num,
                      const T *h_boxes,
                      const T *h_scores,
                      const int64_t *parent_ref_index,
                      const int64_t *parent_ref_count,
                      const T *mean_boxes,
                      const T *mean_scores,
                      T *variances) {
    for (int i = 0; i < parent_object_num; i++) {
        int i_id = PARENT_INDEX(parent_ref_index[i]);

        T inv_N = 1.0 / (static_cast<float>(parent_ref_count[i_id]) - 1.0);
        inv_N = isinf(inv_N) ? 0.0f : inv_N;

        const auto boxes = *reinterpret_cast<const vec4<T> *>(&h_boxes[i * 4]);
        const auto mean = *reinterpret_cast<const vec4<T> *>(&mean_boxes[i_id * 4]);
        const vec4<T> tmp = {mean.x - static_cast<T>(0.5) * (boxes.x + boxes.z),
                             mean.y - static_cast<T>(0.5) * (boxes.y + boxes.w),
                             mean.z - (boxes.z - boxes.x),
                             mean.w - (boxes.w - boxes.y)};

        const vec4<T> var_box = {tmp.x * tmp.x * inv_N,
                                 tmp.y * tmp.y * inv_N,
                                 tmp.z * tmp.z * inv_N,
                                 tmp.w * tmp.w * inv_N};
        const T tmp_delta = mean_scores[i_id] - h_scores[i];
        const T var_score = tmp_delta * tmp_delta * inv_N;

        i_id *= 5;

        variances[i_id + 0] += var_box.x;
        variances[i_id + 1] += var_box.y;
        variances[i_id + 2] += var_box.z;
        variances[i_id + 3] += var_box.w;
        variances[i_id + 4] += var_score;
    }
}

#endif

std::vector <at::Tensor> nms_var_forward(
        const at::Tensor &dets,
        const at::Tensor &scores,
        float nms_overlap_thresh,
        unsigned long top_k) {

    static int iteration = 0;

    TORCH_CHECK(dets.is_cuda(), "dets must be a CUDA tensor")
    TORCH_CHECK(scores.is_cuda(), "scores must be a CUDA tensor")

    TORCH_CHECK(
            dets.dim() == 2, "boxes should be a 2d tensor, got ", dets.dim(), "D")
    TORCH_CHECK(
            dets.size(1) == 4,
            "boxes should have 4 elements in dimension 1, got ",
            dets.size(1))
    TORCH_CHECK(
            scores.dim() == 1,
            "scores should be a 1d tensor, got ",
            scores.dim(),
            "D")
    TORCH_CHECK(
            dets.size(0) == scores.size(0),
            "boxes and scores should have same number of elements in ",
            "dimension 0, got ",
            dets.size(0),
            " and ",
            scores.size(0))

    at::cuda::CUDAGuard device_guard(dets.device());

    if (dets.numel() == 0) {
        std::cout << "returned empty output on iteration " << iteration << std::endl;
        iteration++;
        return {at::empty({0}, dets.options().dtype(at::kLong)),
                at::empty({0}, dets.options().dtype(at::kFloat))};
    }

    auto idx = std::get<1>(scores.sort(/*stable=*/true, /*dim=*/0, /* descending=*/true));

    int dets_num = dets.size(0);

    const int col_blocks = DIVUP(dets_num, threadsPerBlock);

    AT_ASSERTM(col_blocks < MAX_COL_BLOCKS,
               "The number of column blocks must be less than MAX_COL_BLOCKS. Increase the MAX_COL_BLOCKS constant if needed.");

    auto longOptions = torch::TensorOptions().device(torch::kCUDA).dtype(torch::kLong);
    auto mask = at::empty({dets_num * col_blocks}, dets.options().dtype(at::kLong));
    auto keep = at::zeros({dets_num}, longOptions);
    auto parent_ref_index = at::zeros({dets_num}, longOptions);
    auto parent_ref_count = at::zeros({dets_num}, longOptions);
    auto num_to_keep = at::empty({}, longOptions);

    dim3 blocks(col_blocks, col_blocks);
    dim3 threads(threadsPerBlock);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES(
            dets.scalar_type(), "nms_cuda_forward", ([&] {
                nms_map_impl<scalar_t><<<blocks, threads, 0, stream>>>(
                        dets_num,
                        (scalar_t) nms_overlap_thresh,
                        dets.data_ptr<scalar_t>(),
                        idx.data_ptr<int64_t>(),
                        mask.data_ptr<int64_t>());
            }));

    nms_reduce_impl<<<1, 1, 0, stream>>>(dets_num, col_blocks, top_k,
                                         mask.data_ptr<int64_t>(),
                                         idx.data_ptr<int64_t>(),
                                         keep.data_ptr<int64_t>(),
                                         parent_ref_index.data_ptr<int64_t>(),
                                         parent_ref_count.data_ptr<int64_t>(),
                                         num_to_keep.data_ptr<int64_t>());

#ifdef COMPUTE_MEAN_VAR_GPU
    // Reshape this to a [num_to_keep, 4] tensor
    auto parent_object_mean = torch::zeros({num_to_keep.item<int>() * 4},
                                           torch::TensorOptions().device(torch::kCUDA).dtype(torch::kFloat));
    auto parent_scores_mean = torch::zeros({num_to_keep.item<int>() * 1},
                                           torch::TensorOptions().device(torch::kCUDA).dtype(torch::kFloat));
    auto parent_object_var = torch::zeros({num_to_keep.item<int>() * 5},
                                          torch::TensorOptions().device(torch::kCUDA).dtype(torch::kFloat));


    blocks = {static_cast<unsigned int>(DIVUP(parent_ref_index.size(0), threadsPerBlockLinear)), 1, 1};
    threads = {threadsPerBlockLinear, 1, 1};

    AT_DISPATCH_FLOATING_TYPES(dets.scalar_type(), "nms_mean_impl", ([&] {
        nms_mean_impl<scalar_t><<<blocks, threads, 0, stream>>>(parent_ref_index.size(0),
                                                     dets.data_ptr<scalar_t>(),
                                                     scores.data_ptr<scalar_t>(),
                                                     parent_ref_index.data_ptr<int64_t>(),
                                                     parent_ref_count.data_ptr<int64_t>(),
                                                     parent_object_mean.data_ptr<scalar_t>(),
                                                     parent_scores_mean.data_ptr<scalar_t>());
    }));

    AT_DISPATCH_FLOATING_TYPES(dets.scalar_type(), "nms_var_impl", ([&] {
        nms_var_impl<scalar_t><<<blocks, threads, 0, stream>>>(parent_ref_index.size(0),
                                                    dets.data_ptr<scalar_t>(),
                                                    scores.data_ptr<scalar_t>(),
                                                    parent_ref_index.data_ptr<int64_t>(),
                                                    parent_ref_count.data_ptr<int64_t>(),
                                                    parent_object_mean.data_ptr<scalar_t>(),
                                                    parent_scores_mean.data_ptr<scalar_t>(),
                                                    parent_object_var.data_ptr<scalar_t>());
    }));
#else

    // Reshape this to a [num_to_keep, 4] tensor
    auto parent_object_mean = torch::zeros({num_to_keep.item<int>() * 4},
                                           torch::TensorOptions().device(torch::kCPU).dtype(torch::kFloat));
    auto parent_scores_mean = torch::zeros({num_to_keep.item<int>() * 1},
                                           torch::TensorOptions().device(torch::kCPU).dtype(torch::kFloat));
    auto parent_object_var = torch::zeros({num_to_keep.item<int>() * 5},
                                          torch::TensorOptions().device(torch::kCPU).dtype(torch::kFloat));

    auto h_dets = dets.to(torch::kCPU);
    auto h_scores = scores.to(torch::kCPU);
    auto h_parent_ref_index = parent_ref_index.to(torch::kCPU);
    auto h_parent_ref_count = parent_ref_count.to(torch::kCPU);

    nms_mean_impl_cpu(parent_ref_index.size(0),
                      h_dets.data_ptr<float>(),
                      h_scores.data_ptr<float>(),
                      h_parent_ref_index.data_ptr<int64_t>(),
                      h_parent_ref_count.data_ptr<int64_t>(),
                      parent_object_mean.data_ptr<float>(),
                      parent_scores_mean.data_ptr<float>());

    nms_var_impl_cpu(parent_ref_index.size(0),
                     h_dets.data_ptr<float>(),
                     h_scores.data_ptr<float>(),
                     h_parent_ref_index.data_ptr<int64_t>(),
                     h_parent_ref_count.data_ptr<int64_t>(),
                     parent_object_mean.data_ptr<float>(),
                     parent_scores_mean.data_ptr<float>(),
                     parent_object_var.data_ptr<float>());

#endif

    std::cout << "iteration: " << iteration << ", info: " << std::endl;
    std::cout << "dets: " << dets.size(0) << ", scores: " << scores.size(0) << ", keep: " << num_to_keep.item<int>() << std::endl;
    iteration++;

    AT_CUDA_CHECK(hipGetLastError());

    return {keep,//keep.narrow(/*dim=*/0, /*start=*/0, /*length=*/num_to_keep.item<int>()),
            parent_object_var.view({num_to_keep.item<int>(), 5}).to(torch::kCUDA, parent_object_var.scalar_type())};
}
